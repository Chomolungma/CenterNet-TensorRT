#include "hip/hip_runtime.h"
#include "gpu_common.cuh"
#include "custom.hpp"
#include <vector>
#include <cassert>
#include <iostream>

using namespace std;

#define B_BLOCK_VAR_NUMS 256
#define B_ELEM_PT 4 // each thread read 4 var. first

#define M_BLOCK_VAR_NUMS 256
#define M_ELEM_PT 1

const int B_THREADS_PER_BLOCK = B_BLOCK_VAR_NUMS / B_ELEM_PT;

/// bitonic-sort
template <typename T>
__forceinline__ __device__ void compAndSwap(T* data, const size_t i , const size_t j, const bool dir) {
    if (dir == (data[i] > data[j])) {
        T tmp = data[i];
        data[i] = data[j];
        data[j] = tmp;
    }
}

template <typename T>
__forceinline__ __device__ void compAndSwapIndices(T* data, size_t* indices, 
        const size_t i, const size_t j, const bool dir) {
    if (dir == (data[i] > data[j])) {
        T tmp = data[i];
        data[i] = data[j];
        data[j] = tmp;
        size_t idx = indices[i];
        indices[i] = indices[j];
        indices[j] = idx;
    }
}



template <typename T1, typename T2>
__forceinline__ __device__ void set_data(T1& , T2);

template <>
__forceinline__ __device__ void set_data(float& data, double value) {
    data = static_cast<float>(value);
}

template <>
__forceinline__ __device__ void set_data(double& data, double value) {
    data = static_cast<double>(value);
}

template <>
__forceinline__ __device__ void set_data(int& data, double value) {
    data = static_cast<int>(value);
}

template <>
__forceinline__ __device__ void set_data(Pair<float, size_t>& data, double value) {
    data.k = static_cast<float>(value);
    data.v = INT_MAX;
}




#define B2G(x, i) {compAndSwap(x, i, i+1, ascending);}
#define B4G(x, i) { for(int j = 0; j < 2; ++j) { compAndSwap(x, i+j, j+i+2, ascending); } \
    B2G(x, i)  B2G(x, i+2) }

#define B8G(x, i) { for(int j = 0; j < 4; ++j) { compAndSwap(x, i+j, i+j+4, ascending); } \
    B4G(x, i)  B4G(x, i+4) }

#define B16G(x, i) { for(int j = 0; j < 8; ++j) { compAndSwap(x, i+j, i+j+8, ascending); } \
    B8G(x, i) B8G(x, i+8) }

#define B32G(x, i) { for(int j = 0; j < 16; ++j) { compAndSwap(x, i+j, i+j+16, ascending); } \
    B16G(x, i) B16G(x, i+16) } 

#define B64G(x, i) { for(int j = 0; j < 32; ++j)  { compAndSwap(x, i+j, i+j+32, ascending);} \
    B32G(x, i) B32G(x, i+32) }

#define B128G(x, i) { for(int j = 0; j < 64; ++j) compAndSwap(x, i+j, i+j+64, ascending);\
    B64G(x, i) B64G(x, i+64) }

#define B256G(x, i) { for(int j = 0; j < 128; ++j) compAndSwap(x, i+j, i+j+128, ascending);\
    B128G(x, i) B128G(x, i+128) }

#define B2GI(x, k, i) {compAndSwapIndices(x, k, i, i+1, ascending);}
#define B4GI(x, k, i) { for(int j = 0; j < 2; ++j) { compAndSwapIndices(x, k, i+j, j+i+2, ascending); } \
    B2GI(x, k, i)  B2GI(x, k, i+2) }

#define B8GI(x,k, i) { for(int j = 0; j < 4; ++j) { compAndSwapIndices(x, k, i+j, i+j+4, ascending); } \
    B4GI(x, k, i)  B4GI(x, k, i+4) }

#define B16GI(x,k, i) { for(int j = 0; j < 8; ++j) { compAndSwapIndices(x, k, i+j, i+j+8, ascending); } \
    B8GI(x, k, i) B8GI(x, k, i+8) }

#define B32GI(x,k, i) { for(int j = 0; j < 16; ++j) { compAndSwapIndices(x, k, i+j, i+j+16, ascending); } \
    B16GI(x, k, i) B16GI(x, k, i+16) } 

#define B64GI(x, k, i) { for(int j = 0; j < 32; ++j)  { compAndSwapIndices(x, k, i+j, i+j+32, ascending);} \
    B32GI(x, k, i) B32GI(x, k, i+32) }

#define B128GI(x, k, i) { for(int j = 0; j < 64; ++j) compAndSwapIndices(x, k, i+j, i+j+64, ascending);\
    B64GI(x, k, i) B64GI(x, k, i+64) }

#define B256GI(x, k, i) { for(int j = 0; j < 128; ++j) compAndSwapIndices(x, k, i+j, i+j+128, ascending);\
    B128GI(x, k, i) B128GI(x, k, i+128) }


template <typename T>
__global__ void bitonicBlockSort(T* data, const size_t num, const bool sort_asceding) {
    const size_t tid = threadIdx.x;
    const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ T smem[B_BLOCK_VAR_NUMS]; // each block process 256 var.
   
    for(int i = 0; i < B_ELEM_PT; ++i) {
        if (gid*B_ELEM_PT+i < num) smem[tid*B_ELEM_PT + i] = data[gid*B_ELEM_PT + i];
        else {
            if (sort_asceding) set_data(smem[tid*B_ELEM_PT+i], INT_MAX*1.0);
            else set_data(smem[tid*B_ELEM_PT+i], INT_MIN*1.0);
        }
    }
    __syncthreads();
    ///volatile 
    T* sdata = smem + tid * B_ELEM_PT;
    /// 4-group
    bool ascending = sort_asceding;
    B2G(sdata, 0)
    ascending ^= 1;
    B2G(sdata, 2)
    ascending ^= 1;
    __syncthreads();
    if (tid % 2 == 0) { //8-group
        B4G(sdata, 0)
        ascending ^= 1;
        B4G(sdata, 4)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 4 == 0) { // 16-group
        B8G(sdata, 0)
        ascending ^= 1;
        B8G(sdata, 8)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 8 == 0) { // 32-group
        B16G(sdata, 0)
        ascending ^= 1;
        B16G(sdata, 16)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 16 == 0) { //64-group
        B32G(sdata, 0)
        ascending ^= 1;
        B32G(sdata, 32)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 32 == 0) { //128-group
        B64G(sdata, 0)
        ascending ^= 1;
        B64G(sdata, 64)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 64 == 0) { // 256-group
        B128G(sdata, 0)
        ascending ^= 1;
        B128G(sdata, 128)
        ascending ^= 1;
    }
    __syncthreads();
    /// merge
    if (tid == 0) B256G(sdata, 0)
    __syncthreads();
    /// write to global mem.
    for(int i = 0; i < B_ELEM_PT; ++i) {
        data[gid*B_ELEM_PT + i] = sdata[i];
    }
}

template <typename T>
__global__ void bitonicBlockSortIndices(T* data, size_t* indices, const size_t num, const bool sort_asceding) {
    const size_t tid = threadIdx.x;
    const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ T smem[B_BLOCK_VAR_NUMS]; // each block process B_BLOCK_VAR_NUMS var.
    __shared__ size_t sind[B_BLOCK_VAR_NUMS];
   
    for(int i = 0; i < B_ELEM_PT; ++i) {
        if (gid*B_ELEM_PT+i < num) {
            smem[tid*B_ELEM_PT + i] = data[gid*B_ELEM_PT + i];
            sind[tid*B_ELEM_PT + i] = gid*B_ELEM_PT + i;
        }
        else {
            if (sort_asceding) set_data(smem[tid*B_ELEM_PT+i], INT_MAX*1.0);
            else set_data(smem[tid*B_ELEM_PT+i], INT_MIN*1.0);
            sind[tid*B_ELEM_PT+i] = 0;
        }
    }
    __syncthreads();
    ///volatile 
    T* sdata = smem + tid * B_ELEM_PT;
    size_t* sidx  = sind + tid * B_ELEM_PT; 
    /// 4-group
    bool ascending = sort_asceding;
    B2GI(sdata, sidx, 0)
    ascending ^= 1;
    B2GI(sdata, sidx, 2)
    ascending ^= 1;
    __syncthreads();
    if (tid % 2 == 0) { //8-group
        B4GI(sdata, sidx, 0)
        ascending ^= 1;
        B4GI(sdata, sidx, 4)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 4 == 0) { // 16-group
        B8GI(sdata, sidx, 0)
        ascending ^= 1;
        B8GI(sdata, sidx, 8)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 8 == 0) { // 32-group
        B16GI(sdata, sidx, 0)
        ascending ^= 1;
        B16GI(sdata, sidx, 16)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 16 == 0) { //64-group
        B32GI(sdata, sidx, 0)
        ascending ^= 1;
        B32GI(sdata, sidx, 32)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 32 == 0) { //128-group
        B64GI(sdata, sidx, 0)
        ascending ^= 1;
        B64GI(sdata,  sidx, 64)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 64 == 0) { // 256-group
        B128GI(sdata, sidx, 0)
        ascending ^= 1;
        B128GI(sdata, sidx, 128)
        ascending ^= 1;
    }
    __syncthreads();
    /// merge
    if (tid == 0) B256GI(sdata, sidx, 0)
    __syncthreads();

    for(int i = 0; i < B_ELEM_PT; ++i) {
        data[gid*B_ELEM_PT + i] = sdata[i];
        indices[gid*B_ELEM_PT+i] = sidx[i];
    }
}

template <typename T>
__global__ void bitonicBatchBlockSortIndices(T* data, size_t* indices, const int batch_num,
        const size_t slice_len, const size_t padding_len, 
        const bool sort_asceding) {
    const size_t tid = threadIdx.x;
    const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;

    size_t g_addr, t_addr;
    size_t index;
    __shared__ T smem[B_BLOCK_VAR_NUMS]; // each block process 256 var.
    __shared__ size_t sind[B_BLOCK_VAR_NUMS];

    const size_t batch_id = gid * B_ELEM_PT / padding_len;
    const size_t shift = batch_id * padding_len;

    for(int i = 0; i < B_ELEM_PT; ++i) {
        g_addr = gid * B_ELEM_PT + i;
        t_addr = tid * B_ELEM_PT + i;
        index  = g_addr - shift;
        if (index < slice_len ) {
            smem[t_addr] = data[g_addr];
            sind[t_addr] = index;
        }
        else {
            if (sort_asceding) set_data(smem[t_addr], INT_MAX*1.0);
            else set_data(smem[t_addr], INT_MIN*1.0);
            sind[t_addr] = 0;
        }
    }
    __syncthreads();
    //volatile 
    T* sdata = smem + tid * B_ELEM_PT;
    //volatile 
    size_t* sidx  = sind + tid * B_ELEM_PT; 
    /// 4-group
    bool ascending = sort_asceding;
    B2GI(sdata, sidx, 0)
    ascending ^= 1;
    B2GI(sdata, sidx, 2)
    ascending ^= 1;
    __syncthreads();
    if (tid % 2 == 0) { //8-group
        B4GI(sdata, sidx, 0)
        ascending ^= 1;
        B4GI(sdata, sidx, 4)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 4 == 0) { // 16-group
        B8GI(sdata, sidx, 0)
        ascending ^= 1;
        B8GI(sdata, sidx, 8)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 8 == 0) { // 32-group
        B16GI(sdata, sidx, 0)
        ascending ^= 1;
        B16GI(sdata, sidx, 16)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 16 == 0) { //64-group
        B32GI(sdata, sidx, 0)
        ascending ^= 1;
        B32GI(sdata, sidx, 32)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 32 == 0) { //128-group
        B64GI(sdata, sidx, 0)
        ascending ^= 1;
        B64GI(sdata,  sidx, 64)
        ascending ^= 1;
    }
    __syncthreads();
    if (tid % 64 == 0) { // 256-group
        B128GI(sdata, sidx, 0)
        ascending ^= 1;
        B128GI(sdata, sidx, 128)
        ascending ^= 1;
    }
    __syncthreads();
    /// merge
    if (tid == 0) B256GI(sdata, sidx, 0)
    __syncthreads();

    for(int i = 0; i < B_ELEM_PT; ++i) {
        data[gid*B_ELEM_PT + i] = sdata[i];
        indices[gid*B_ELEM_PT+i] = sidx[i];
    }
}

/// merge-sort
template <typename T>
__host__ __device__ void merge(T* left, const size_t left_len, T* right, const size_t right_len,
        T* dest, bool up) {
    size_t i = 0, j = 0, k = 0;
    while(i < left_len && j < right_len) {
        if(!up) {
            if(left[i] > right[j]) dest[k++] = left[i++];
            else dest[k++] = right[j++];
        } else {
            if(left[i] < right[j]) dest[k++] = left[i++];
            else dest[k++] = right[j++];
        }
    }
    while( i < left_len ) dest[k++] = left[i++];
    while( j < right_len ) dest[k++] = right[j++];
}
 
template <typename T>
__global__ void merge_sort_inplace(T* in, const size_t num, bool up) {

    const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t tid = threadIdx.x;
    /// 256 threads per block
    __shared__ T smem[256];
    __shared__ T sout[256]; 

    if (gid < num) smem[tid] = in[gid];
    else {
        if (up) set_data(smem[tid], INT_MAX*1.0);
        else set_data(smem[tid], INT_MIN*1.0);
    }
    __syncthreads();

    if (tid < 128) merge<T>(smem+tid, 1, smem+(tid+128), 1, sout+2*tid, up);
    __syncthreads();
    if (tid < 64) merge<T>(sout+tid*2, 2, sout+(tid+64)*2, 2, smem+4*tid, up);
    __syncthreads();
    if (tid < 32) merge<T>(smem+tid*4, 4, smem+(tid+32)*4, 4, sout+8*tid, up);
    __syncthreads();
    if (tid < 16) merge<T>(sout+tid*8, 8, sout+(tid+16)*8, 8, smem+16*tid, up);
    __syncthreads();
    if (tid < 8) merge<T>(smem+tid*16, 16, smem+(tid+8)*16, 16, sout+32*tid, up);
    __syncthreads();
    if (tid < 4) merge<T>(sout+tid*32, 32, sout+(tid+4)*32, 32, smem+64*tid, up);
    __syncthreads();
    if (tid < 2) merge<T>(smem+tid*64, 64, smem+(tid+2)*64, 64, sout+128*tid, up);
    __syncthreads();
    if (tid < 1) merge<T>(sout+tid*128, 128, sout+(tid+1)*128, 128, smem+256*tid, up);
    __syncthreads();

    in[gid] = smem[tid];
    __syncthreads();
}


template <typename T>
__global__ void merge_blocks_result(T* data, 
        const size_t seg_len, const size_t mid,  
        T* out, const bool up) {
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(mid == 0) return;
    merge<T>(data + seg_len*tid, seg_len, data+seg_len*(tid+mid), seg_len, out+seg_len*2*tid, up);
}


template <typename T>
__device__ void topk_merge_two_blocks(T* left, size_t* i_left,
        T* right, size_t* i_right, const int K) {
    int i, j;
    T tmp;
    size_t i_tmp;

    if (left[K-1] > right[0]) return;
    if (left[0] < right[K-1]) {
        for(i = 0; i < K; ++i) { left[i] = right[i]; i_left[i] = i_right[i]; }
        return;
    }
    for( i = 0; i < K; ++i ) {
        if (left[i] > right[0]) continue;
        tmp = left[i];
        i_tmp = i_left[i];
        left[i] = right[0];
        i_left[i] = i_right[0];
        for(j = 1; j < K; ++j) {
            if (tmp < right[j]) {
                right[j-1] = right[j];
                i_right[j-1] = i_right[j];
            } else {
                right[j-1] = tmp; 
                i_right[j-1] = i_tmp;
                break;
            } 
        }
    }
}



template <typename T>
__global__ void topk_merge_blocks(T* data,  size_t* indices, 
        const int  block_var_num, 
        const int slice_blocks_num, 
        const int padding_blocks_num, 
        const int batch_num,  
        const int K) {
    const size_t tid = threadIdx.x; 
    const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t batch_id = gid / padding_blocks_num;
    const size_t valid_id = batch_id * padding_blocks_num + slice_blocks_num;

    size_t r_shift ;
    size_t threads = blockDim.x ;
    /// merge within a block
    while (threads > 1 && tid < (threads >> 1) ) {
        r_shift = gid + (threads>>1) ;
        if (r_shift < valid_id)  { // valid mem. access
            topk_merge_two_blocks<T>(data + block_var_num * gid, indices + block_var_num * gid,
                                 data + block_var_num * r_shift, indices + block_var_num * r_shift, K);
        }
        threads >>= 1;
        __syncthreads();
    }
}


template <typename T>
__global__ void align_mem(T* data, size_t* indices, const int batch_num, 
        const size_t slice_blocks_num, const size_t block_var_num, const int K) {
    if (threadIdx.x == 0) {
        for (int i = 1; i < batch_num; ++i)  {
            for( int j = 0; j < K; ++j ) {
                data[K * i + j] = data[slice_blocks_num*block_var_num*i + j];
                indices[K * i + j] = indices[slice_blocks_num * block_var_num * i + j];
            }
        }
    }
}


template <typename T1, typename T2>
T1 divUp(T1 a, T2 b) {
    return (a + b - 1) / b;
}


template <typename T>
__global__ void merge_seg_kernel(T* in, const size_t num, const int block_var_num,  
        size_t* vol, size_t* shift, const int param_size, T* out, bool up) {
    T v ;
    int k = 0;

    size_t pt [32];
    size_t idx;
    for(int i  = 0; i < param_size; ++i) pt[i] = 0;

    for(size_t m = 0; m < num; ++m) {
        if(up) set_data(v, INT_MAX * 1.0);
        else set_data(v, INT_MIN * 1.0);
        for(int i = 0; i < param_size; ++i) {
            idx = shift[i] * block_var_num + pt[i];
            if (up && pt[i] < vol[i] * block_var_num && idx < num && v > in[idx]) {
                v = in[idx];
                k = i;
            }
            if (!up && pt[i] < vol[i] * block_var_num && idx < num && v < in[idx]) {
                v = in[idx];
                k = i;
            } 
        }
        out[m] = in[shift[k] * block_var_num + pt[k]];
        ++pt[k];
    }
}




template <typename T>
void merge_sorted_arrays(T* in, const size_t num, T* buff, const bool up,
        const int merge_blocks_num, const int block_var_num) {
    /// merge blocks' result
    vector<int> param;
    log2_series(merge_blocks_num, param);

    DoubleBuffer<T> buffers(in, buff);
    size_t seg_len = block_var_num;
    size_t sum = 0;
    size_t shift = 0;
    size_t threads_per_block, num_blocks;
    for(size_t i = 0; i < param.size(); ++i) {
        if (!param[i])  break;
        size_t vol = 2 << (param[i] - 1); 
        sum += vol;
        int threads = vol >> 1; 
        while(threads) {
            threads_per_block = threads > block_var_num ? block_var_num: threads;
            num_blocks = threads > block_var_num ? divUp(threads, threads_per_block) : 1;
            merge_blocks_result<T><<<num_blocks, threads_per_block>>>(buffers.Current(), seg_len, threads, buffers.Alternate(), up);
            buffers.selector = buffers.selector ^ 1;
            threads >>= 1; 
            seg_len <<= 1;
        }
        if (param.size() == 1)  {
            if(buffers.Current() == in) return;
            hipMemcpy(in, buffers.Current(), sizeof(T)*num, hipMemcpyDeviceToDevice);
            return; 
        }
        if (buffers.Current() != buff + shift) {
            hipMemcpy(buff + shift, buffers.Current(), sizeof(T) * vol * block_var_num, hipMemcpyDeviceToDevice);
        }
        shift += vol * block_var_num;
        buffers = DoubleBuffer<T>(in + shift, buff + shift); 
        seg_len = block_var_num;
    }
    if (!param.back()) {
        hipMemcpy(buff+shift, in+sum*block_var_num, sizeof(T) * block_var_num, hipMemcpyDeviceToDevice);
    }
    /// merge multiple sorted arrays in GPU
    size_t h_vol[32];
    size_t h_shift[32];
    for(size_t i = 0; i < param.size(); ++i) {
        h_vol[i] = param[i] > 0 ? 2 << (param[i] - 1) : 1;
        h_shift[i] = (i == 0) ? 0 : h_shift[i-1] + h_vol[i-1]; 
    }

    size_t* d_vol;
    size_t* d_shift;
    hipMalloc((void**)&d_vol, sizeof(size_t) * param.size());
    hipMalloc((void**)&d_shift, sizeof(size_t) * param.size());

    hipMemcpy(d_vol, h_vol, sizeof(size_t) * param.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_shift, h_shift, sizeof(size_t) * param.size(), hipMemcpyHostToDevice);

    merge_seg_kernel<T><<<1, 1>>>(buff, num, block_var_num, d_vol, d_shift, param.size(), in, up);

    hipFree(d_vol);
    hipFree(d_shift);
}


template <typename T>
void merge_batch_topk(T* idata, size_t* indices, const int batch_num, const size_t padding_len,  
        const int K, const int slice_blocks_num, const int block_var_num,
        hipStream_t stream) {
    /// blocks: |0, 1, ..., slice_blocks_num-1 | ... | ... |

    const int threads_per_block = 128;

    int blocks_per_batch = slice_blocks_num;
    int padding_blocks_num = divUp(slice_blocks_num, threads_per_block) * threads_per_block;
    int num_blocks = padding_blocks_num / threads_per_block * batch_num;
    size_t num_per_block = block_var_num;

    /// merge within each block
    while (blocks_per_batch > 1) {
        topk_merge_blocks<T><<<num_blocks, threads_per_block, 0, stream>>>(idata, indices, num_per_block,  \
                blocks_per_batch, padding_blocks_num, batch_num, K);
        num_per_block *= threads_per_block;
        blocks_per_batch = num_blocks/batch_num;
        padding_blocks_num = divUp(blocks_per_batch, threads_per_block) * threads_per_block;
        num_blocks = padding_blocks_num / threads_per_block * batch_num;
    }
}


template <typename T>
void mergeSort(T* in, const size_t num, T* buff,  const bool up) {
    const int threads_per_block = M_BLOCK_VAR_NUMS;
    const int num_blocks = divUp(num, threads_per_block);
    /// sort the sub-arrays
    merge_sort_inplace<T><<<num_blocks, threads_per_block>>>(
            in, num,  up);

    /// merge 
    merge_sorted_arrays<T>(in, num, buff, up, num_blocks, M_BLOCK_VAR_NUMS);
}

template <typename T>
void bitonicSort(T* in, const size_t num, T* buff, const bool ascending) {
    int num_blocks = (num + B_BLOCK_VAR_NUMS - 1) / B_BLOCK_VAR_NUMS;
    /// inplace sort within each block
    bitonicBlockSort<T><<<num_blocks, B_THREADS_PER_BLOCK>>>(in, num, ascending);
    /// merge
    merge_sorted_arrays<T>(in, num, buff, ascending, num_blocks, B_BLOCK_VAR_NUMS);
}




template <typename T>
void bitonicBatchTopK(T* data, size_t* indices, const int batch_num, 
        const size_t slice_len, const int K,
        hipStream_t stream) {

    ///inplace sort with blocks
    size_t padding_len = (slice_len + B_BLOCK_VAR_NUMS - 1) / B_BLOCK_VAR_NUMS * B_BLOCK_VAR_NUMS;
    int num_blocks = padding_len / B_BLOCK_VAR_NUMS * batch_num;  

    bitonicBatchBlockSortIndices<T><<<num_blocks, B_THREADS_PER_BLOCK, 0, stream>>>(data, indices, batch_num, slice_len, padding_len, false);
    /// merge
    merge_batch_topk<T>(data, indices, batch_num, padding_len, K, num_blocks / batch_num, B_BLOCK_VAR_NUMS, stream);
    if (batch_num > 1) {
        align_mem<T><<<1, 1, 0, stream>>>(data, indices, batch_num,  num_blocks/batch_num, B_BLOCK_VAR_NUMS, K);
    }
}

__global__ void ctdet_decode_kernel(
        float* det, 
        float* scores, size_t* indices, float* wh, float* reg,  
        float* trans,
        const int batch_num, const size_t slice_blocks_num, 
        const size_t block_var_num, const int K, 
        const int height, const int width, 
        const bool reg_exist, 
        const int num_joints, const float thresh
        ) {
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= batch_num * K) return;
    if (scores[tid] < thresh) return;  //no nms, directly using threshold

    const size_t area  = height * width;
    const size_t batch_id = tid / K;
    const size_t local_id = tid % K;

    const size_t iid = slice_blocks_num * block_var_num * batch_id + local_id;
    const int batch_len = 1 + 6 * K;  
    
    size_t class_id = indices[iid] / area;
    indices[iid] %= area;

    atomicAdd(&det[batch_id * batch_len], 1.0);

    float bias = 0.;
    float xs, ys;
    if (!reg_exist) bias = 0.5;
    
    ys = static_cast<size_t>(indices[iid] / width) * 1.0 + bias;
    xs = static_cast<size_t>(indices[iid] % width) * 1.0 + bias;


    if (reg_exist) { // reg: Nx2xHxW -> Nx2xK
        xs += reg[batch_id*2*area + indices[iid]];
        ys += reg[batch_id*2*area + area + indices[iid]];
    }
    float wh1 = wh[batch_id*2*area + indices[iid]] / 2.0;
    float wh2 = wh[batch_id*2*area + area + indices[iid]] / 2.0;

    float t0, t1, t2, t3;
    float tt0, tt1, tt2, tt3;
    t0 = xs - wh1;
    t1 = ys - wh2;
    t2 = xs + wh1;
    t3 = ys + wh2;

    /// inverse-warpAffine
    tt0 = trans[0] * t0 + trans[1] * t1 + trans[2]; 
    tt1 = trans[3] * t0 + trans[4] * t1 + trans[5];
    tt2 = trans[0] * t2 + trans[1] * t3 + trans[2];
    tt3 = trans[3] * t2 + trans[4] * t3 + trans[5];

    //printf("id:%d, score:%.4f, cls:%d, box:(%.1f, %.1f, %.1f, %.1f)\n", tid,  scores[tid], class_id, tt0, tt1, tt2, tt3);
    /// det: N* (1 + 6*K)
    
    det[batch_id * batch_len + local_id * 6 + 0 + 1] = class_id;
    det[batch_id * batch_len + local_id * 6 + 1 + 1] = scores[tid];
    det[batch_id * batch_len + local_id * 6 + 2 + 1] = tt0;
    det[batch_id * batch_len + local_id * 6 + 3 + 1] = tt1;
    det[batch_id * batch_len + local_id * 6 + 4 + 1] = tt2;
    det[batch_id * batch_len + local_id * 6 + 5 + 1] = tt3;

}

__global__ void pose_decode_kernel(
        float* det, 
        float* scores, size_t* indices, float* wh, float* reg,  
        float* hps, float* hm_hp, size_t* hm_ind, 
        float* hp_offset, 
        float* trans,
        const int batch_num, const size_t slice_blocks_num, 
        const size_t block_var_num, const int K, 
        const int height, const int width, 
        const bool reg_exist, const bool hm_hp_exist, 
        const int num_joints, const float thresh
        ) {
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= batch_num * K) return;
    if (scores[tid] < thresh) return;  //! no nms, directly using threshold

    const int res_num = 2 + 4 + num_joints * 2;
    const int batch_len = (1 + res_num * K);

    const size_t area  = height * width;
    const size_t batch_id = tid / K;
    const size_t local_id = tid % K;
    atomicAdd(&det[batch_id * batch_len], 1.0); //! number of det

    const size_t iid = slice_blocks_num * block_var_num * batch_id + local_id;
    
    size_t class_id = indices[iid] / area;
    indices[iid] %= area;

    float xs, ys;
    float p0, p1;
    float t0, t1, t2, t3;
    float tt0, tt1, tt2, tt3;
    float bias[2];
    ys = static_cast<size_t>(indices[iid] / width) * 1.0;
    xs = static_cast<size_t>(indices[iid] % width) * 1.0;

    if (reg_exist) { // reg: Nx2xHxW -> Nx2xK
        bias[0] = reg[batch_id*2*area + indices[iid]];
        bias[1] = reg[batch_id*2*area + area + indices[iid]];
    } else {
        bias[0] = 0.5; bias[1] = 0.5;
    }
    float wh1 = wh[batch_id*2*area + indices[iid]] / 2.0;
    float wh2 = wh[batch_id*2*area + area + indices[iid]] / 2.0;

    t0 = xs - wh1 + bias[0];
    t1 = ys - wh2 + bias[1];
    t2 = xs + wh1 + bias[0];
    t3 = ys + wh2 + bias[1];

    /// inverse-warpAffine
    tt0 = trans[0] * t0 + trans[1] * t1 + trans[2]; 
    tt1 = trans[3] * t0 + trans[4] * t1 + trans[5];
    tt2 = trans[0] * t2 + trans[1] * t3 + trans[2];
    tt3 = trans[3] * t2 + trans[4] * t3 + trans[5];
    det[batch_id * batch_len + local_id * res_num + 0 + 1] = class_id;
    det[batch_id * batch_len + local_id * res_num + 1 + 1] = scores[tid];
    det[batch_id * batch_len + local_id * res_num + 2 + 1] = tt0;
    det[batch_id * batch_len + local_id * res_num + 3 + 1] = tt1;
    det[batch_id * batch_len + local_id * res_num + 4 + 1] = tt2;
    det[batch_id * batch_len + local_id * res_num + 5 + 1] = tt3;

    /// key points
    for(int i = 0; i < num_joints; ++i) {
        p0 = hps[batch_id*num_joints*2*area + i*2*area + indices[iid]] + xs;
        p1 = hps[batch_id*num_joints*2*area + (i*2+1)*area + indices[iid]] + ys;
        
        /// find the most closed point with a confidence > 0.1 
        if (hm_hp_exist) {
            float min_ds = static_cast<float>(INT_MAX);
            float near_xs = min_ds, near_ys = min_ds;

            // hm_hp: N x 17 x 128 x 128 
            float hm_hp_score, diff = min_ds;
            float hm_xs, hm_ys;
            size_t ind_tmp;
            for(int j = 0; j < K; ++j) {
                hm_hp_score = hm_hp[batch_id * num_joints * area + i * area + j];
                if (hm_hp_score < 0.1)  continue;
                ind_tmp = hm_ind[batch_id*num_joints*area + i * area + j] % area;
                hm_ys = static_cast<size_t>(ind_tmp / width) * 1.0 + hp_offset[batch_id*2*area + area + j];
                hm_xs = static_cast<size_t>(ind_tmp % width) * 1.0 + hp_offset[batch_id*2*area + j];
                diff = fabs(p0 - hm_xs) + fabs(p1 - hm_ys);
                if (diff < min_ds) {
                    min_ds = diff;
                    near_xs = hm_xs;
                    near_ys = hm_ys;
                }
            }
            if (near_xs > t0 && near_xs < t2 && near_ys > t1 &&
                    near_ys < t3 && diff < max(t2-t0, t3-t1) * 0.5) {
                p0 = near_xs;
                p1 = near_ys;
            }
        }
        
        tt0 = trans[0] * p0 + trans[1] * p1 + trans[2];
        tt1 = trans[3] * p0 + trans[4] * p1 + trans[5];

        det[batch_id * batch_len + local_id * res_num + i*2 + 7] = tt0;
        det[batch_id * batch_len + local_id * res_num + i*2+1 + 7] = tt1;
    }
}

void ctdet_decode(
        float* det,  
        float* wh, float* reg, 
        float* heat, size_t* indices,    
        float* inv_trans, 
        const int batch_num, const int num_classes,  
        const int height, const int width, 
        const int K, const float threshold, 
        const bool reg_exist, const bool cat_spec_wh,
        hipStream_t stream) {

    ///inplace sort with blocks
    const size_t slice_len = height * width * num_classes;
    const size_t padding_len = (slice_len + B_BLOCK_VAR_NUMS - 1) / B_BLOCK_VAR_NUMS * B_BLOCK_VAR_NUMS;
    int num_blocks = padding_len / B_BLOCK_VAR_NUMS * batch_num;  

    bitonicBatchBlockSortIndices<float><<<num_blocks, B_THREADS_PER_BLOCK, 0, stream>>>(heat, indices, batch_num, slice_len, padding_len, false);
    CHECK_LAST_ERR("ctdet_bitonic_sort_kernel");
    /// merge
    merge_batch_topk<float>(heat, indices, batch_num, padding_len, K, num_blocks / batch_num, B_BLOCK_VAR_NUMS, stream);
    CHECK_LAST_ERR("ctdet_merge_batch_topk_kernel");
    ///
    ctdet_decode_kernel<<<divUp(K * batch_num, 128), 128, 0, stream>>>(
            det, heat, indices, 
            wh, reg, inv_trans,    
            batch_num, num_blocks/batch_num, 
            B_BLOCK_VAR_NUMS, K, height, width, reg_exist,
            num_classes, threshold);

    CHECK_LAST_ERR("ctdet_decode_kernel");
}


void multi_pose_decode(
        float* det,  
        float* heat, float* wh, float* reg, 
        float* hps, float* hm_hp, float* hp_offset, 
        size_t* heat_ind, size_t* hm_ind, 
        float* inv_trans, 
        const int batch_num, const int num_joints, 
        const int height, const int width, 
        const int K, const float threshold, 
        const bool reg_exist, const bool hm_hp_exist, 
        hipStream_t stream) {

    const size_t area = height * width;
    const size_t heat_slice_len = area * 1; 
    const size_t heat_padding_len = (heat_slice_len + B_BLOCK_VAR_NUMS - 1) / B_BLOCK_VAR_NUMS * B_BLOCK_VAR_NUMS;
    const int heat_num_blocks = heat_padding_len / B_BLOCK_VAR_NUMS * batch_num;  

    const size_t hm_slice_len = area;
    const size_t hm_padding_len = (area + B_BLOCK_VAR_NUMS-1)/B_BLOCK_VAR_NUMS * B_BLOCK_VAR_NUMS;
    const int hm_batch_num = batch_num * num_joints;
    const int hm_num_blocks = hm_padding_len / B_BLOCK_VAR_NUMS * batch_num * num_joints;
    
    /// get the Top-K of the heat map
    bitonicBatchBlockSortIndices<float><<<heat_num_blocks, B_THREADS_PER_BLOCK, 0, stream>>>(heat, heat_ind, batch_num, heat_slice_len, heat_padding_len, false);
    CHECK_LAST_ERR("heat_bitonic_sort_kernel");

    merge_batch_topk<float>(heat, heat_ind, batch_num, heat_padding_len, K, heat_num_blocks / batch_num, B_BLOCK_VAR_NUMS, stream);
    CHECK_LAST_ERR("heat_merge_batch_topk_kernel");

    /// get the channel Top-K of hm_hp
    if (hm_hp_exist) {
        bitonicBatchBlockSortIndices<float><<<hm_num_blocks, B_THREADS_PER_BLOCK, 0, stream>>>(hm_hp, hm_ind, hm_batch_num, \
                hm_slice_len, hm_padding_len, false);
        CHECK_LAST_ERR("hm_bitonic_sort_kernel");
        merge_batch_topk<float>(hm_hp, hm_ind, hm_batch_num, hm_padding_len, K, hm_num_blocks / hm_batch_num, B_BLOCK_VAR_NUMS, stream);
        CHECK_LAST_ERR("heat_merge_batch_topk_kernel");
    }

    /// decode 
    pose_decode_kernel<<<divUp(K * batch_num, 128), 128, 0, stream>>>(
            det, heat, heat_ind, 
            wh, reg, hps, 
            hm_hp, hm_ind, 
            hp_offset, inv_trans,    
            batch_num, heat_num_blocks/batch_num, 
            B_BLOCK_VAR_NUMS, K, height, width, 
            reg_exist, hm_hp_exist, 
            num_joints, threshold);

    CHECK_LAST_ERR("pose_decode_kernel");
}

template void mergeSort<int>(int* , const size_t , int*,  const bool);
template void mergeSort<float>(float*, const size_t, float*,  const bool);
template void mergeSort<double>(double*, const size_t, double*,  const bool);
template void mergeSort<Pair<float, size_t>>(Pair<float, size_t>*, const size_t, Pair<float, size_t>*, const bool);


template void bitonicSort<int>(int *, const size_t, int*, const bool);
template void bitonicSort<float>(float *, const size_t, float*, const bool);
template void bitonicSort<double>(double *, const size_t, double*, const bool);
template void bitonicSort<Pair<float, size_t>>(Pair<float, size_t>*, const size_t, Pair<float, size_t>*, const bool);


template void bitonicBatchTopK<int>(int*, size_t*, const int, const size_t, const int, hipStream_t);
template void bitonicBatchTopK<float>(float*, size_t*, const int, const size_t, const int, hipStream_t);
template void bitonicBatchTopK<double>(double*, size_t*, const int, const size_t, const int, hipStream_t);

