#include "hip/hip_runtime.h"
/*
 * This file contains some kernels including the pre-processing, post-processing of ctdet. 
 * 
 */
#include <math.h>
#include "det_kernels.hpp"
#include <iostream>

#include "gpu_common.cuh"
#include "custom.hpp"



#ifdef USE_CV_WARP_AFFINE //! cv::warpAffine is slower than GPU's implementation

template <typename T>
__device__ __forceinline__ static T area_pixel_compute_source_index(T scale,
                                     int dst_index,
                                     bool align_corners,
                                     bool cubic = false) {
    if (align_corners) {
        return scale * dst_index;
    }
    else {
        T src_idx = scale * (dst_index + static_cast<T>(0.5)) - static_cast<T>(0.5);
        return (!cubic && src_idx < static_cast<T>(0)) ? static_cast<T>(0): src_idx;
    }
}

template <typename T>
__host__ __forceinline__ T area_pixel_compute_scale(int input_size, int output_size, bool align_corners) {
    if(output_size > 1) {
        return align_corners ? static_cast<T>(input_size - 1) / (output_size - 1) : static_cast<T>(input_size) / output_size;
    }
    else {
        return static_cast<T>(0);
    }
}


/*
 * in_img: NHWC, out_img: NCHW (may be normalized), mean/std: gpu mem. 
 *  C == 3 
 */ 
__global__ void preprocess_kernel(const int batch_size, 
        const uint8_t* in_img, const int channels, const int in_h, const int in_w, 
        float* out_img, const int out_h, const int out_w, 
        const float rheight, const float rwidth, const bool align_corners, 
        const float padding_val, 
        const ScaleOp type, 
        const float* mean, const bool mean_valid, 
        const float* std, const bool std_valid) {

    /// 2D Index of current thread
    size_t threadX = blockIdx.x * blockDim.x + threadIdx.x;
    size_t threadY = blockIdx.y * blockDim.y + threadIdx.y;
    size_t batch_id = static_cast<int>(threadY / out_h);
    int yIndex = threadY % out_h; 
    int xIndex = threadX;
    if(xIndex >= out_w || threadY >= out_h * batch_size) return;
    const size_t out_pannel = out_h * out_w * channels;  


    if (in_h != out_h || in_w != out_w) {  
        if (type == ScaleOp::Resize) { //! resize using bilinear interpolation 
            float h1r = area_pixel_compute_source_index<float>(rheight, yIndex, align_corners, /*cubic=*/false);
            const int h1 = h1r;
            const int h1p = (h1 < in_h - 1) ? 1 : 0;
            const float h1lambda = h1r - h1;
            const float h0lambda = static_cast<float>(1) - h1lambda;
            //
            float w1r = area_pixel_compute_source_index<float>(rwidth, xIndex, align_corners, /*cubic=*/false);
            const int w1 = w1r;
            const int w1p = (w1 < in_w - 1) ? 1 : 0;
            const float w1lambda = w1r - w1;
            const float w0lambda = static_cast<float>(1) - w1lambda;
            //
            for (int c = 0; c < channels; ++c) {
                const int outIdx = c * out_h * out_w + yIndex * out_w + xIndex + batch_id * out_pannel; // NCHW
                const float val = 
                    h0lambda * (w0lambda * in_img[batch_id*out_pannel + h1 * in_w * channels + w1 * channels + channels - 1 - c] +
                     w1lambda * in_img[batch_id*out_pannel + h1 * in_w * channels + (w1 + w1p) * channels +  channels - 1 - c]) +
                    h1lambda *
                    (w0lambda * in_img[batch_id * out_pannel + (h1 + h1p) * in_w * channels + w1 * channels + channels - 1 - c] +
                     w1lambda * in_img[batch_id * out_pannel + (h1 + h1p) * in_w * channels + (w1 + w1p) * channels + channels -1 -c]);
                out_img[outIdx] = val / 255.0; 
               if(mean_valid) out_img[outIdx] -= mean[c];
               if(std_valid)  out_img[outIdx] /= std[c];
            } 
        } else if (type == ScaleOp::Padding) { //! padding, center aligned 
            const int x1_x = out_w > in_w ? (out_w - in_w) / 2 : 0;
            const int x1_y = out_h > in_h ? (out_h - in_h) / 2 : 0;
            for(int c = 0; c < channels; ++c) {
                const int outIdx = c * out_h * out_w + yIndex * out_w + xIndex + batch_id * out_pannel;
                if (yIndex >= x1_y && yIndex < x1_y + in_h && 
                        xIndex >= x1_x && xIndex < x1_x + in_w) {
                    const int inIdx = (yIndex - x1_y) * in_w * channels + (xIndex - x1_x) * channels + batch_id * out_pannel; 
                    out_img[outIdx] = static_cast<float>(in_img[inIdx + channels - 1 - c]) / 255.0; // BGR -> RGB
                    if(mean_valid) out_img[outIdx] -= mean[c];
                    if(std_valid)  out_img[outIdx] /= std[c];
                } else {
                    out_img[outIdx] = padding_val;  
                }
            }
        }
    } else { //! same shape
        const int idx = batch_id * out_pannel +  
                yIndex * in_w * channels + xIndex * channels; // NHWC
        for(int c = 0; c < channels; ++c) {
            const int outIdx = c * out_h * out_w + yIndex * out_w + \
			       xIndex + batch_id * out_pannel; //NCHW
            //out_img[outIdx] = static_cast<float>(in_img[idx + channels - 1 - c]) / 255.0; // BGR -> RGB
            out_img[outIdx] = static_cast<float>(in_img[idx + c]) / 255.0;  // keep BGR format
            if(mean_valid) out_img[outIdx] -= mean[c];
            if(std_valid)  out_img[outIdx] /= std[c];
        }
    }
}




/*
 * 1. read image first
 * 2. allocate the gpu mem. of gpu_mat & inp_img
 * 3. affineTransform
 * 4. BGR HWC -> RGB CHW & normalization
 */ 
void cuda_preprocess(const int batch_size,
        float* d_out, 
        const uint8_t* gpu_mat, \
        const cv::Mat& img, 
        cv::Mat& inp_img, 
        const int input_h, const int input_w, 
        const float scale, const int pad, \
        const bool fix_res, const float down_ratio, \
        float* inv_trans, 
        const float* mean, const bool mean_valid,
        const float* std, const bool std_valid, 
        hipStream_t& stream) {
    int new_height = static_cast<int>(img.rows * scale);
    int new_width  = static_cast<int>(img.cols * scale);
    int inp_h, inp_w;
    float c[2], s[2];
    if(fix_res) {
        inp_h = input_h;
        inp_w = input_w;
        c[0] = new_width / 2.;
        c[1] = new_height / 2.;
        s[0] = img.rows > img.cols ? img.rows : img.cols;
        s[1] = s[0];
    } else {
        inp_h = (new_height | pad) + 1;
        inp_w = (new_width | pad) + 1;
        c[0] = new_width / 2 ;
        c[1] = new_height / 2;
        s[0] = inp_w;
        s[1] = inp_h;
    }
    /// affine_transform
    float shift[2] = {0., 0.};
    using namespace cv;
    cv::Mat warp_mat (2, 3, CV_32FC1);
    get_affine_transform((float*)warp_mat.data, c, s, shift, 0, inp_h, inp_w);
    get_affine_transform(inv_trans, c, s, shift, 0, inp_h / down_ratio, inp_w / down_ratio, true);

    if (new_width != img.cols && new_height != img.rows) {
        cv::resize(img, img, cv::Size(new_width, new_height));
    }
    warpAffine(img, inp_img, warp_mat, inp_img.size());
    //cv::imwrite("warp2.png", inp_img);

    /// copy to gpu memory 
    CHECK_CUDA(hipMemcpyAsync((void*)gpu_mat, inp_img.data, sizeof(uint8_t)*inp_img.rows*inp_img.cols*3, hipMemcpyHostToDevice, stream));
    /// BGR HWC -> RGB, CHW & normalization
    dim3 block(16, 16);
    int grid_x = (inp_w + block.x - 1) / block.x;
    int grid_y = (inp_h*batch_size + block.y - 1) / block.y;
    dim3 grid(grid_x, grid_y);
    // bool align_corners = true;
    // float rheight = area_pixel_compute_scale<float>(inp_h, inp_h, align_corners);
    // float rwidth  = area_pixel_compute_scale<float>(inp_w, inp_w, align_corners);

    preprocess_kernel<<<grid, block, 0, stream>>>(
	        batch_size, 
            static_cast<const uint8_t* >(gpu_mat), 
            3, inp_h, inp_w,  
            d_out, inp_h, inp_w, 
            1, 1, true, 
            0, 
            ScaleOp::Same, 
            static_cast<const float* >(mean), mean_valid, 
            static_cast<const float* >(std), std_valid);

    CHECK_LAST_ERR("preprocess_kernel");
}
#else
/// warpAffine: src -> dst, since the traversal locates on dst rather than src,
/// we need the inverse transformation matrix
/// dst = trans * src, src = inv_trans * dst  

/// src: NHWC, dst:NCHW, keep BGR format
template <typename T>
__global__ void centernet_preprocess_kernel(const int batch_num, 
        T* src,  const int channel, 
        const int in_h, const int in_w, 
        float* dst, const int out_h, const int out_w, 
        const float* inv_trans,
        const float* mean, const bool mean_valid,
        const float* std,  const bool std_valid) {

    const int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t threadY = threadIdx.y + blockIdx.y * blockDim.y;
    if (xIndex >= out_w || threadY >= out_h * batch_num) return;
    const int yIndex = threadY % out_h; 
    const int batch_id = threadY / out_h; 
    const size_t out_pannel = out_h * out_w * channel;
    const size_t in_pannel = in_h * in_w * channel;

    /// calculate the indices of src.
    const float sx = inv_trans[0] * static_cast<float>(xIndex) + inv_trans[1] * static_cast<float>(yIndex) + inv_trans[2];
    const float sy = inv_trans[3] * static_cast<float>(xIndex) + inv_trans[4] * static_cast<float>(yIndex) + inv_trans[5];

    float val = 0.;

    const int h1 = sy;
    const int h1p = (h1 < in_h - 1) ? 1 : 0;

    const float h1lambda = sy - h1 * 1.0;
    const float h0lambda = 1.0  - h1lambda;
    //
    const int w1 = sx;
    const int w1p = (w1 < in_w - 1) ? 1 : 0;

    const float w1lambda = sx - w1 ;
    const float w0lambda = 1.0 - w1lambda;
    size_t outIdx; 
    const float d_mean[3] = {0.408, 0.447, 0.470};
    const float d_std[3]  = {0.289, 0.274, 0.278};
    //
    for (int c = 0; c < channel; ++c) {
        //outIdx = c  + yIndex * out_w * channel + xIndex * channel + batch_id * out_pannel; // NHWC
        outIdx = c * out_h * out_w + yIndex * out_w + xIndex + batch_id * out_pannel; // NCHW
        if (sx < 0 || sx > in_w || sy < 0|| sy > in_h) {
            val = 0;
        } else { //! bilinear interpolation 
            val = h0lambda * (w0lambda * src[batch_id*in_pannel + h1 * in_w * channel + w1 * channel + c] + 
                 w1lambda * src[batch_id* in_pannel + h1 * in_w * channel + (w1 + w1p) * channel +  c]) +
                h1lambda * (w0lambda * src[batch_id * in_pannel + (h1 + h1p) * in_w * channel + w1 * channel + c] +
                 w1lambda * src[batch_id * in_pannel + (h1 + h1p) * in_w * channel + (w1 + w1p) * channel + c]);
        }

        /*
        dst[outIdx] /= 255.0;
        if (mean_valid) {
            dst[outIdx] -= mean[c];
        }
        if (std_valid) {
            dst[outIdx] /= std[c];
        }
        */
        dst[outIdx] = (val / 255.0 - d_mean[c] ) / d_std[c];
    } 
}






template <typename T>
void cuda_centernet_preprocess(const int batch_num, 
        T* src, const int channel, const int in_h, const int in_w,  
        float* dst, const int out_h, const int out_w, 
        const float* inv_trans,  
        const float* mean, const bool mean_valid,
        const float* std, const bool std_valid, 
        hipStream_t stream) {

    dim3 block(16, 16);
    int grid_x = (out_w + block.x - 1) / block.x;
    int grid_y = (out_h*batch_num + block.y - 1) / block.y;
    dim3 grid(grid_x, grid_y);

    centernet_preprocess_kernel<T><<<grid, block, 0, stream>>>(batch_num,
            src, channel, in_h, in_w, dst, out_h, out_w, 
            inv_trans, mean, mean_valid, std, std_valid);
}

template void cuda_centernet_preprocess(const int, uint8_t*, const int, const int, const int, \
        float*, const int, const int, const float*, const float*, const bool, \
        const float*, const bool, hipStream_t);


#endif 
